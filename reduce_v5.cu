#include <hip/hip_runtime.h>

#include <iostream>
#include <numeric>

#define BLOCK_SIZE 1024
// 使用 shared memory
// 但是v2中线程数优化失效，因为每个线程至少要负责读取一个元素
__global__ void reduce_sum_v5(float *input, float *output, int n) {
    __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        sdata[tid] = input[i];
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride && i + stride < n) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(output, sdata[0]);
    }
}

float *generate_data(int n) {
    srand(time(NULL)); // time.h 包含在了 cuda_runtime.h 中
    float *data = new float[n];
    for (int i = 0; i < n; i++) {
        data[i] = rand() % 10;
    }
    return data;
}

int main() {
    int n = 1 << 20;
    float *input_h = generate_data(n);
    float *output_h = new float;

    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    float *input_d, *output_d;
    hipMalloc(&input_d, n * sizeof(float));
    hipMalloc(&output_d, sizeof(float));

    hipMemcpy(input_d, input_h, n * sizeof(float), hipMemcpyHostToDevice);

    reduce_sum_v5<<<numBlocks, BLOCK_SIZE>>>(input_d, output_d, n);

    hipMemcpy(output_h, output_d, sizeof(float), hipMemcpyDeviceToHost);
    
    // c++17 特性
    float sum = std::reduce(input_h, input_h + n);
    std::cout << "sum (cpu): " << sum << std::endl;
    std::cout << "sum (gpu): " << *output_h << std::endl;

    if (sum == *output_h) {
        std::cout << "Test passed!" << std::endl;
    } else {
        std::cout << "Test failed!" << std::endl;
    }

    delete[] input_h;
    delete output_h;
    hipFree(input_d);
    hipFree(output_d);

    return 0;
}