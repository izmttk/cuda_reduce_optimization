#include <hip/hip_runtime.h>

#include <iostream>
#include <numeric>

// Interleaved Addressing 交错寻址，即把具有相同条件的线程相邻放置（或者说放在同一个 warp 中）
// 优化 warp divergence
__global__ void reduce_sum_v3(float *input, float *output, int n) {
    // 在每个 block 内进行独立的 reduce
    int size = blockDim.x * 2;
    int offset = blockIdx.x * size;
    int tid = threadIdx.x;
    for (int stride = 1; stride < size; stride *= 2) {
        // 每个线程负责一次 reduce
        int i = tid * 2 * stride; // 被加和元素的块内位置
        if (i < size && offset + i + stride < n) {
            input[offset + i] += input[offset + i + stride];
        }
        __syncthreads();
    }
    if (tid == 0) {
        atomicAdd(output, input[offset]);
    }
}

float *generate_data(int n) {
    srand(time(NULL));
    float *data = new float[n];
    for (int i = 0; i < n; i++) {
        data[i] = rand() % 10;
    }
    return data;
}

int main() {
    int n = 1 << 20;
    float *input_h = generate_data(n);
    float *output_h = new float;

    int blockSize = 1024;
    // numBlocks = ceil((n / 2) / blockSize) = ceil(n / (blockSize * 2))
    int numBlocks = (n + blockSize * 2 - 1) / (blockSize * 2);

    float *input_d, *output_d;
    hipMalloc(&input_d, n * sizeof(float));
    hipMalloc(&output_d, sizeof(float));

    hipMemcpy(input_d, input_h, n * sizeof(float), hipMemcpyHostToDevice);

    reduce_sum_v3<<<numBlocks, blockSize>>>(input_d, output_d, n);

    hipMemcpy(output_h, output_d, sizeof(float), hipMemcpyDeviceToHost);
    
    // c++17 特性
    float sum = std::reduce(input_h, input_h + n);
    std::cout << "sum (cpu): " << sum << std::endl;
    std::cout << "sum (gpu): " << *output_h << std::endl;

    if (sum == *output_h) {
        std::cout << "Test passed!" << std::endl;
    } else {
        std::cout << "Test failed!" << std::endl;
    }

    delete[] input_h;
    delete output_h;
    hipFree(input_d);
    hipFree(output_d);

    return 0;
}