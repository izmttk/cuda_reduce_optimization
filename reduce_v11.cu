#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>

// pytorch 的 block reduce 写法
#define WARP_SIZE 32
#define MAX_BLOCK_SIZE 1024

struct Sum {
    template<typename T>
    __device__ __forceinline__ T operator()(T a, T b) const {
        return a + b;
    }
};

struct Max {
    template<typename T>
    __device__ __forceinline__ T operator()(T a, T b) const {
        return a > b ? a : b;
    }
};

struct Min {
    template<typename T>
    __device__ __forceinline__ T operator()(T a, T b) const {
        return a < b ? a : b;
    }
};

template<typename T>
__host__ __device__ __forceinline__ constexpr T div_ceiling(T a, T b) {
    return (a + b - 1) / b;
}

// 一个 warp 有 32 个线程
// 全部 32 个线程都会被 reduce，不判断边界
template<typename T, typename Op = Sum, const int WarpSz = WARP_SIZE>
__device__ __forceinline__ T warp_reduce(T val, Op op = Op()) {
#pragma unroll
    for (int offset = WarpSz / 2 ; offset > 0; offset /= 2) {
        val = op(val, __shfl_down_sync(0xffffffff, val, offset));
    }
    return val;
}

// 一个 block 最多 1024 个线程，即 32 个 warp
template<typename T, typename Op = Sum, const int BlockSz = MAX_BLOCK_SIZE, const int WarpSz = WARP_SIZE>
__device__ __forceinline__ T block_reduce(T val, T* sdata, Op op = Op(), const T ident = T(0)) {
    const int tid = threadIdx.x;
    const int warp = tid / WarpSz;
    const int lane = tid % WarpSz;
    constexpr int maxNumWarps = div_ceiling(BlockSz, WarpSz);

    val = warp_reduce(val, op);
    // __syncthreads();
    // 如果要连续调用 block_reduce，需要在此处或者两个函数调用之间同步
    // 否则如果 sdata 相同，会发生读写冲突，举例如下：
    // sum1 = BlockReduceSum(val1, shared);
    // sum2 = BlockReduceSum(val2, shared);
    // 1、假设有block中有2个warp，在第二次__syncthreads()之后，warp1执行较快结束进入第二个BlockReduceSum中
    // 2、此时 warp0 正准备读取shared[1]，但warp1先在第二个BlockReduceSum中写入shared[1]，这就出现了读写冲突
    if (lane == 0) { // 这里会发生 warp divergence，无法避免
        // 为什么在 ncu 中这里会观察到少量 bank conflict？
        sdata[warp] = val; // 写共享内存
    }
    // 同样在共享内存读写之间需要同步
    // 否则第1个 warp 在写完共享内存后开始读取了，其他 warp 还没写完
    __syncthreads();
    val = (tid < maxNumWarps) ? sdata[tid] : ident; // 读共享内存
    if (warp == 0) {
        val = warp_reduce<T, Op, maxNumWarps>(val, op);
    }
    return val;
}

// Reference:
// https://github.com/pytorch/pytorch/blob/042f2f7746a064f1527d95d1f1d712b4f0b34186/aten/src/ATen/native/cuda/Reduce.cuh#L689
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#memory-fence-functions

template<typename T, typename Op, const int BlockSz, const int WarpSz>
__device__ __forceinline__ void reduce(T *input, T *output, int n, Op op, const T ident, T *block_output, int *counter) {
    constexpr int maxNumWarps = div_ceiling(BlockSz, WarpSz);
    __shared__ T sdata[maxNumWarps];

    int tid = threadIdx.x;

    T val = ident;

    int numThreads = gridDim.x * blockDim.x; // 网格中所有线程数

    // 反正最大并行度摆在这了，直接把输入规模缩减到 gridDim.x * blockDim.x，理论计算时间应该是不变的
    int numTiles = div_ceiling(n, numThreads);
    int input_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // 循环展开在这里没什么用
    for (int tile = 0; tile < numTiles; tile++) {
        int idx = input_idx + tile * numThreads;
        if (idx < n) {
            val = op(val, input[idx]);
        }
    }
    
    // 这里不需要 __syncthreads()，因为 warp_reduce 中的 __*sync 函数
    // 保证 warp 内所有被 mask 选中的线同步才会继续执行
    val = block_reduce<T, Op, BlockSz, WarpSz>(val, sdata, op, ident);

    __shared__ bool is_last_block_done; // 注意该变量全 block 共享
    if (tid == 0) {
        block_output[blockIdx.x] = val;

        // 接下来是 block 同步机制
        
        // 首先确保 count 的写入发生在 output 写入之后、
        // 这样只要执行了 atomicAdd，就表示 output 写入操作已经完成
        __threadfence();

        // 每有一个 block 完成，count + 1
        int prev_block_finished = atomicAdd(counter, 1);
        // 所有 block 完成后 prev_block_finished == gridDim.x - 1
        is_last_block_done = (prev_block_finished == gridDim.x - 1);
    }
    // is_last_block_done 由每个块的第一个线程写入
    // 防止其他线程在第一个线程写入之前读取
    __syncthreads();
    // 最后一步由最后一个写入 output 的 block 完成
    if (is_last_block_done) {
        val = (tid < gridDim.x) ? block_output[tid] : ident;
        val = block_reduce<T, Op, BlockSz, WarpSz>(val, sdata, op, ident);
        if (tid == 0) {
            output[0] = val;
        }
    }
}

// 仅仅是一个包装函数
template<typename T, typename Op, const int BlockSz, const int WarpSz>
__global__ void reduce_kernel(T *input, T *output, int n, Op op, const T ident, T *block_output, int *counter) {
    reduce<T, Op, BlockSz, WarpSz>(input, output, n, op, ident, block_output, counter);
}

template<typename T, typename Op>
void launch_reduce_kernel(T *input, T *output, int n, Op op, T ident) {
    constexpr int blockSize = 512;
    constexpr int warpSize = WARP_SIZE;

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    // GPU占用率 = active warps / supported maximum active warps
    int maxActiveBlocks =  deviceProp.maxThreadsPerMultiProcessor / blockSize * deviceProp.multiProcessorCount;
    printf("maxThreadsPerMultiProcessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
    printf("multiProcessorCount: %d\n", deviceProp.multiProcessorCount);
    printf("maxActiveBlocks: %d\n", maxActiveBlocks);
    // 让block数不超过最大活动block数
    int numBlocks = min(maxActiveBlocks, div_ceiling(n, blockSize));
    // int numBlocks = div_ceiling(n, blockSize);
    T *block_output;
    int *counter;
    hipMalloc(&block_output, numBlocks * sizeof(T));
    hipMalloc(&counter, sizeof(int));
    hipMemset(counter, 0, sizeof(int));
    printf("Launch config: grid(%d, 1, 1), block(%d, 1, 1)\n", numBlocks, blockSize);
    reduce_kernel<T, Op, blockSize, warpSize><<<numBlocks, blockSize>>>(input, output, n, op, ident, block_output, counter);
    hipFree(block_output);
    hipFree(counter);
}

template<typename T>
void reduce_sum(T *input, T *output, int n) {
    Sum op = Sum();
    T ident = T(0);
    launch_reduce_kernel(input, output, n, op, ident);
}

template<typename T>
void reduce_max(T *input, T *output, int n) {
    Max op = Max();
    T ident = std::numeric_limits<T>::min();
    launch_reduce_kernel(input, output, n, op, ident);
}

template<typename T>
void reduce_min(T *input, T *output, int n) {
    Min op = Min();
    T ident = std::numeric_limits<T>::max();
    launch_reduce_kernel(input, output, n, op, ident);
}

template<typename T>
T *generate_data(int n) {
    srand(time(NULL)); // time.h 包含在了 hip/hip_runtime.h 中
    T *data = new T[n];
    for (int i = 0; i < n; i++) {
        data[i] = rand() % 100;
    }
    return data;
}

using Type = float; // 测试类型，请注意浮点类型会因为精度问题导致测试失败

int main() {
    int n = 0;
    std::cout << "Input n: ";
    std::cin >> n;
    // 一些重要的输入规模
    // 只有一个 block： 1 << 4 + 1 = 17
    // 多于一个 block 但是小于最大并行 block 数：1 << 11 + 1 = 2049
    // 多余最大并行 block 数：1 << 22 + 1 = 4194305
    Type *input_h = generate_data<Type>(n);
    Type *output_h = new Type;

    Type *input_d, *output_d;
    hipMalloc(&input_d, n * sizeof(Type));
    hipMalloc(&output_d, sizeof(Type));

    hipMemcpy(input_d, input_h, n * sizeof(Type), hipMemcpyHostToDevice);

    // test sum
    reduce_sum(input_d, output_d, n);
    hipMemcpy(output_h, output_d, sizeof(Type), hipMemcpyDeviceToHost);
    Type sum = std::reduce(input_h, input_h + n);
    std::cout << "sum (cpu): " << sum << std::endl;
    std::cout << "sum (gpu): " << *output_h << std::endl;

    if (sum == *output_h) {
        std::cout << "Test passed!" << std::endl;
    } else {
        std::cout << "Test failed!" << std::endl;
    }

    // test max
    reduce_max(input_d, output_d, n);
    hipMemcpy(output_h, output_d, sizeof(Type), hipMemcpyDeviceToHost);
    Type max = std::reduce(input_h, input_h + n, std::numeric_limits<Type>::min(), [](Type a, Type b) { return a > b ? a : b; });
    std::cout << "max (cpu): " << max << std::endl;
    std::cout << "max (gpu): " << *output_h << std::endl;

    if (max == *output_h) {
        std::cout << "Test passed!" << std::endl;
    } else {
        std::cout << "Test failed!" << std::endl;
    }

    // test min
    reduce_min(input_d, output_d, n);
    hipMemcpy(output_h, output_d, sizeof(Type), hipMemcpyDeviceToHost);
    Type min = std::reduce(input_h, input_h + n, std::numeric_limits<Type>::max(), [](Type a, Type b) { return a < b ? a : b; });    // c++17 特性
    std::cout << "min (cpu): " << min << std::endl;
    std::cout << "min (gpu): " << *output_h << std::endl;

    if (min == *output_h) {
        std::cout << "Test passed!" << std::endl;
    } else {
        std::cout << "Test failed!" << std::endl;
    }

    delete[] input_h;
    delete output_h;
    hipFree(input_d);
    hipFree(output_d);

    return 0;
}