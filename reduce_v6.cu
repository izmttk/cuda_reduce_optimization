#include <hip/hip_runtime.h>

#include <iostream>
#include <numeric>

// 使用 shared memory
// 同时减少空闲线程数
__global__ void reduce_sum_v6(float *input, float *output, int n) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    if (i + blockDim.x < n) {
        sdata[tid] = input[i] + input[i + blockDim.x];
    } else if (i < n) {
        sdata[tid] = input[i];
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride && i + stride < n) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(output, sdata[0]);
    }
}

float *generate_data(int n) {
    srand(time(NULL)); // time.h 包含在了 cuda_runtime.h 中
    float *data = new float[n];
    for (int i = 0; i < n; i++) {
        data[i] = rand() % 10;
    }
    return data;
}

int main() {
    int n = 1 << 20;
    float *input_h = generate_data(n);
    float *output_h = new float;
    
    int blockSize = 1024;
    int numBlocks = (n + blockSize * 2 - 1) / (blockSize * 2);

    float *input_d, *output_d;
    hipMalloc(&input_d, n * sizeof(float));
    hipMalloc(&output_d, sizeof(float));

    hipMemcpy(input_d, input_h, n * sizeof(float), hipMemcpyHostToDevice);

    reduce_sum_v6<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(input_d, output_d, n);

    hipMemcpy(output_h, output_d, sizeof(float), hipMemcpyDeviceToHost);
    
    // c++17 特性
    float sum = std::reduce(input_h, input_h + n);
    std::cout << "sum (cpu): " << sum << std::endl;
    std::cout << "sum (gpu): " << *output_h << std::endl;

    if (sum == *output_h) {
        std::cout << "Test passed!" << std::endl;
    } else {
        std::cout << "Test failed!" << std::endl;
    }

    delete[] input_h;
    delete output_h;
    hipFree(input_d);
    hipFree(output_d);

    return 0;
}